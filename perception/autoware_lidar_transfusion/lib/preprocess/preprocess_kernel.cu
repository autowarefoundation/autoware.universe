#include "hip/hip_runtime.h"
// Copyright 2024 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "autoware/lidar_transfusion/cuda_utils.hpp"
#include "autoware/lidar_transfusion/preprocess/preprocess_kernel.hpp"

#include <cstdint>

namespace autoware::lidar_transfusion
{

PreprocessCuda::PreprocessCuda(const TransfusionConfig & config, hipStream_t & stream)
: stream_(stream), config_(config)
{
  mask_size_ = config_.grid_z_size_ * config_.grid_y_size_ * config_.grid_x_size_;
  voxels_size_ = config_.grid_z_size_ * config_.grid_y_size_ * config_.grid_x_size_ *
                   config_.max_num_points_per_pillar_ * config_.num_point_feature_size_ +
                 1;
  mask_ = cuda::make_unique<unsigned int[]>(mask_size_);
  voxels_ = cuda::make_unique<float[]>(voxels_size_);
}

__global__ void generateSweepPoints_kernel(
  const uint8_t * input_data, size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  union {
    uint32_t raw{0};
    float value;
  } input_x, input_y, input_z;

#pragma unroll
  for (int i = 0; i < 4; i++) {  // 4 bytes for float32
    input_x.raw |= input_data[point_idx * input_point_step + i] << i * 8;
    input_y.raw |= input_data[point_idx * input_point_step + i + 4] << i * 8;
    input_z.raw |= input_data[point_idx * input_point_step + i + 8] << i * 8;
  }

  float input_intensity = static_cast<float>(input_data[point_idx * input_point_step + 12]);

  output_points[point_idx * num_features] =
    transform_array[0] * input_x.value + transform_array[4] * input_y.value +
    transform_array[8] * input_z.value + transform_array[12];
  output_points[point_idx * num_features + 1] =
    transform_array[1] * input_x.value + transform_array[5] * input_y.value +
    transform_array[9] * input_z.value + transform_array[13];
  output_points[point_idx * num_features + 2] =
    transform_array[2] * input_x.value + transform_array[6] * input_y.value +
    transform_array[10] * input_z.value + transform_array[14];
  output_points[point_idx * num_features + 3] = input_intensity;
  output_points[point_idx * num_features + 4] = time_lag;
}

hipError_t PreprocessCuda::generateSweepPoints_launch(
  const uint8_t * input_data, size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, float * output_points)
{
  dim3 blocks(divup(points_size, config_.threads_for_voxel_));
  dim3 threads(config_.threads_for_voxel_);

  generateSweepPoints_kernel<<<blocks, threads, 0, stream_>>>(
    input_data, points_size, input_point_step, time_lag, transform_array,
    config_.num_point_feature_size_, output_points);

  hipError_t err = hipGetLastError();
  return err;
}

__global__ void shufflePoints_kernel(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= max_size) return;

  int src_idx = indices[(point_idx + offset) % max_size];
  int dst_idx = point_idx;

  if (dst_idx >= points_size) {
    shuffled_points[5 * dst_idx + 0] = INFINITY;
    shuffled_points[5 * dst_idx + 1] = INFINITY;
    shuffled_points[5 * dst_idx + 2] = INFINITY;
    shuffled_points[5 * dst_idx + 3] = INFINITY;
    shuffled_points[5 * dst_idx + 4] = INFINITY;
  } else {
    shuffled_points[5 * dst_idx + 0] = points[5 * src_idx + 0];
    shuffled_points[5 * dst_idx + 1] = points[5 * src_idx + 1];
    shuffled_points[5 * dst_idx + 2] = points[5 * src_idx + 2];
    shuffled_points[5 * dst_idx + 3] = points[5 * src_idx + 3];
    shuffled_points[5 * dst_idx + 4] = points[5 * src_idx + 4];
  }
}

hipError_t PreprocessCuda::shufflePoints_launch(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset)
{
  dim3 blocks(divup(points_size, config_.threads_for_voxel_));
  dim3 threads(config_.threads_for_voxel_);

  if (blocks.x == 0) {
    return hipGetLastError();
  }

  shufflePoints_kernel<<<blocks, threads, 0, stream_>>>(
    points, indices, shuffled_points, points_size, max_size, offset);
  hipError_t err = hipGetLastError();
  return err;
}

void PreprocessCuda::generateVoxels(
  float * points, unsigned int points_size, unsigned int * pillar_num, float * voxel_features,
  unsigned int * voxel_num, unsigned int * voxel_idxs)
{
  cuda::clear_async(mask_.get(), mask_size_, stream_);
  cuda::clear_async(voxels_.get(), voxels_size_, stream_);
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));

  CHECK_CUDA_ERROR(generateVoxels_random_launch(points, points_size, mask_.get(), voxels_.get()));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));

  CHECK_CUDA_ERROR(generateBaseFeatures_launch(
    mask_.get(), voxels_.get(), pillar_num, voxel_features, voxel_num, voxel_idxs));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));
}

__global__ void generateVoxels_random_kernel(
  float * points, unsigned int points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, int points_per_voxel, unsigned int * mask,
  float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float x = points[point_idx * 5];
  float y = points[point_idx * 5 + 1];
  float z = points[point_idx * 5 + 2];
  float i = points[point_idx * 5 + 3];
  float t = points[point_idx * 5 + 4];

  if (
    x <= min_x_range || x >= max_x_range || y <= min_y_range || y >= max_y_range ||
    z <= min_z_range || z >= max_z_range)
    return;

  int voxel_idx = floorf((x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((y - min_y_range) / pillar_y_size);
  voxel_idx = voxel_idx < 0 ? 0 : voxel_idx >= grid_x_size ? grid_x_size - 1 : voxel_idx;
  voxel_idy = voxel_idy < 0 ? 0 : voxel_idy >= grid_y_size ? grid_y_size - 1 : voxel_idy;
  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= points_per_voxel) return;
  float * address = voxels + (voxel_index * points_per_voxel + point_id) * 5;
  atomicExch(address + 0, x);
  atomicExch(address + 1, y);
  atomicExch(address + 2, z);
  atomicExch(address + 3, i);
  atomicExch(address + 4, t);
}

hipError_t PreprocessCuda::generateVoxels_random_launch(
  float * points, unsigned int points_size, unsigned int * mask, float * voxels)
{
  if (points_size == 0) {
    return hipGetLastError();
  }
  dim3 blocks(divup(points_size, config_.threads_for_voxel_));
  dim3 threads(config_.threads_for_voxel_);

  generateVoxels_random_kernel<<<blocks, threads, 0, stream_>>>(
    points, points_size, config_.min_x_range_, config_.max_x_range_, config_.min_y_range_,
    config_.max_y_range_, config_.min_z_range_, config_.max_z_range_, config_.voxel_x_size_,
    config_.voxel_y_size_, config_.voxel_z_size_, config_.grid_y_size_, config_.grid_x_size_,
    config_.points_per_voxel_, mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, float points_per_voxel,
  float max_voxels, unsigned int * pillar_num, float * voxel_features, unsigned int * voxel_num,
  unsigned int * voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (voxel_idx >= grid_x_size || voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < points_per_voxel ? count : points_per_voxel;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId >= max_voxels) return;

  voxel_num[current_pillarId] = count;

  uint4 idx = {0, 0, voxel_idy, voxel_idx};
  ((uint4 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * points_per_voxel + i;
    int outIndex = current_pillarId * points_per_voxel + i;
    voxel_features[outIndex * 5] = voxels[inIndex * 5];
    voxel_features[outIndex * 5 + 1] = voxels[inIndex * 5 + 1];
    voxel_features[outIndex * 5 + 2] = voxels[inIndex * 5 + 2];
    voxel_features[outIndex * 5 + 3] = voxels[inIndex * 5 + 3];
    voxel_features[outIndex * 5 + 4] = voxels[inIndex * 5 + 4];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t PreprocessCuda::generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, unsigned int * pillar_num, float * voxel_features,
  unsigned int * voxel_num, unsigned int * voxel_idxs)
{
  dim3 threads = {32, 32};
  dim3 blocks = {divup(config_.grid_x_size_, threads.x), divup(config_.grid_y_size_, threads.y)};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream_>>>(
    mask, voxels, config_.grid_y_size_, config_.grid_x_size_, config_.points_per_voxel_,
    config_.max_voxels_, pillar_num, voxel_features, voxel_num, voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

}  // namespace autoware::lidar_transfusion
