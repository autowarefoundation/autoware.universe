#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "autoware/lidar_centerpoint/cuda_utils.hpp"
#include "autoware/lidar_centerpoint/preprocess/preprocess_kernel.hpp"
#include "autoware/lidar_centerpoint/utils.hpp"

#include <cassert>
#include <cmath>

namespace
{
const std::size_t MAX_POINT_IN_VOXEL_SIZE = 32;  // the same as max_point_in_voxel_size_ in config
const std::size_t WARPS_PER_BLOCK = 4;
const std::size_t ENCODER_IN_FEATURE_SIZE = 9;  // the same as encoder_in_feature_size_ in config
}  // namespace

namespace autoware::lidar_centerpoint
{

__global__ void generateSweepPoints_kernel(
  const float * input_points, std::size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  const float input_x = input_points[point_idx * input_point_step + 0];
  const float input_y = input_points[point_idx * input_point_step + 1];
  const float input_z = input_points[point_idx * input_point_step + 2];

  // transform_array is expected to be column-major
  output_points[point_idx * num_features + 0] = transform_array[0] * input_x +
                                                transform_array[4] * input_y +
                                                transform_array[8] * input_z + transform_array[12];
  output_points[point_idx * num_features + 1] = transform_array[1] * input_x +
                                                transform_array[5] * input_y +
                                                transform_array[9] * input_z + transform_array[13];
  output_points[point_idx * num_features + 2] = transform_array[2] * input_x +
                                                transform_array[6] * input_y +
                                                transform_array[10] * input_z + transform_array[14];
  output_points[point_idx * num_features + 3] = time_lag;
}

hipError_t generateSweepPoints_launch(
  const float * input_points, std::size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points, hipStream_t stream)
{
  auto transform_d = cuda::make_unique<float[]>(16);
  CHECK_CUDA_ERROR(hipMemcpyAsync(
    transform_d.get(), transform_array, 16 * sizeof(float), hipMemcpyHostToDevice, stream));

  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);
  assert(num_features == 4);

  generateSweepPoints_kernel<<<blocks, threads, 0, stream>>>(
    input_points, points_size, input_point_step, time_lag, transform_d.get(), num_features,
    output_points);

  hipError_t err = hipGetLastError();
  return err;
}

__global__ void shufflePoints_kernel(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= max_size) return;

  int src_idx = indices[(point_idx + offset) % max_size];
  int dst_idx = point_idx;

  if (dst_idx >= points_size) {
    shuffled_points[4 * dst_idx + 0] = INFINITY;
    shuffled_points[4 * dst_idx + 1] = INFINITY;
    shuffled_points[4 * dst_idx + 2] = INFINITY;
    shuffled_points[4 * dst_idx + 3] = INFINITY;
  } else {
    shuffled_points[4 * dst_idx + 0] = points[4 * src_idx + 0];
    shuffled_points[4 * dst_idx + 1] = points[4 * src_idx + 1];
    shuffled_points[4 * dst_idx + 2] = points[4 * src_idx + 2];
    shuffled_points[4 * dst_idx + 3] = points[4 * src_idx + 3];
  }
}

hipError_t shufflePoints_launch(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset,
  hipStream_t stream)
{
  dim3 blocks((max_size + 256 - 1) / 256);
  dim3 threads(256);

  if (blocks.x == 0) {
    return hipGetLastError();
  }

  shufflePoints_kernel<<<blocks, threads, 0, stream>>>(
    points, indices, shuffled_points, points_size, max_size, offset);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateVoxels_random_kernel(
  const float * points, std::size_t points_size, float min_x_range, float max_x_range,
  float min_y_range, float max_y_range, float min_z_range, float max_z_range, float pillar_x_size,
  float pillar_y_size, float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask,
  float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float4 point = ((float4 *)points)[point_idx];

  if (
    point.x < min_x_range || point.x >= max_x_range || point.y < min_y_range ||
    point.y >= max_y_range || point.z < min_z_range || point.z >= max_z_range)
    return;

  int voxel_idx = floorf((point.x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range) / pillar_y_size);
  voxel_idx = voxel_idx < 0 ? 0 : voxel_idx >= grid_x_size ? grid_x_size - 1 : voxel_idx;
  voxel_idy = voxel_idy < 0 ? 0 : voxel_idy >= grid_y_size ? grid_y_size - 1 : voxel_idy;
  unsigned int voxel_index = (grid_x_size - 1 - voxel_idx) * grid_y_size + voxel_idy;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= MAX_POINT_IN_VOXEL_SIZE) return;
  float * address = voxels + (voxel_index * MAX_POINT_IN_VOXEL_SIZE + point_id) * 4;
  atomicExch(address + 0, point.x);
  atomicExch(address + 1, point.y);
  atomicExch(address + 2, point.z);
  atomicExch(address + 3, point.w);
}

hipError_t generateVoxels_random_launch(
  const float * points, std::size_t points_size, float min_x_range, float max_x_range,
  float min_y_range, float max_y_range, float min_z_range, float max_z_range, float pillar_x_size,
  float pillar_y_size, float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask,
  float * voxels, hipStream_t stream)
{
  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);

  if (blocks.x == 0) {
    return hipGetLastError();
  }

  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
    points, points_size, min_x_range, max_x_range, min_y_range, max_y_range, min_z_range,
    max_z_range, pillar_x_size, pillar_y_size, pillar_z_size, grid_y_size, grid_x_size, mask,
    voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs)
{
  // exchange x and y to process in a row-major order
  // flip x axis direction to process front to back
  unsigned int voxel_idx_inverted = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int voxel_idy = blockIdx.x * blockDim.x + threadIdx.x;
  if (voxel_idx_inverted >= grid_x_size || voxel_idy >= grid_y_size) return;
  unsigned int voxel_idx = grid_x_size - 1 - voxel_idx_inverted;

  unsigned int voxel_index = voxel_idx_inverted * grid_y_size + voxel_idy;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < MAX_POINT_IN_VOXEL_SIZE ? count : MAX_POINT_IN_VOXEL_SIZE;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId > max_voxel_size - 1) return;

  voxel_num[current_pillarId] = count;

  uint3 idx = {0, voxel_idy, voxel_idx};
  ((uint3 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * MAX_POINT_IN_VOXEL_SIZE + i;
    int outIndex = current_pillarId * MAX_POINT_IN_VOXEL_SIZE + i;
    ((float4 *)voxel_features)[outIndex] = ((float4 *)voxels)[inIndex];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs,
  hipStream_t stream)
{
  // exchange x and y to process in a row-major order
  dim3 threads = {32, 32};
  dim3 blocks = {
    (grid_y_size + threads.x - 1) / threads.x, (grid_x_size + threads.y - 1) / threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>(
    mask, voxels, grid_y_size, grid_x_size, max_voxel_size, pillar_num, voxel_features, voxel_num,
    voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateFeatures_kernel(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const float voxel_x, const float voxel_y, const float voxel_z,
  const float range_min_x, const float range_min_y, const float range_min_z, float * features)
{
  // voxel_features (float): (max_voxel_size, max_point_in_voxel_size, point_feature_size)
  // voxel_num_points (int): (max_voxel_size)
  // coords (int): (max_voxel_size, point_dim_size)
  int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;
  int point_idx = threadIdx.x % MAX_POINT_IN_VOXEL_SIZE;
  int pillar_idx_inBlock = threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;  // max_point_in_voxel_size

  unsigned int num_pillars = num_voxels[0];
  if (pillar_idx >= num_pillars) return;

  // load src
  __shared__ float4 pillarSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE];
  __shared__ float3 pillarSumSM[WARPS_PER_BLOCK];
  __shared__ int3 cordsSM[WARPS_PER_BLOCK];
  __shared__ int pointsNumSM[WARPS_PER_BLOCK];
  __shared__ float pillarOutSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][ENCODER_IN_FEATURE_SIZE];

  if (threadIdx.x < WARPS_PER_BLOCK) {
    pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    cordsSM[threadIdx.x] = ((int3 *)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    pillarSumSM[threadIdx.x] = {0, 0, 0};
  }

  pillarSM[pillar_idx_inBlock][point_idx] =
    ((float4 *)voxel_features)[pillar_idx * MAX_POINT_IN_VOXEL_SIZE + point_idx];
  __syncthreads();

  // calculate sm in a pillar
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x), pillarSM[pillar_idx_inBlock][point_idx].x);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y), pillarSM[pillar_idx_inBlock][point_idx].y);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z), pillarSM[pillar_idx_inBlock][point_idx].z);
  }
  __syncthreads();

  // feature-mean
  float3 mean;
  float validPoints = pointsNumSM[pillar_idx_inBlock];
  mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
  mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
  mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

  mean.x = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
  mean.y = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
  mean.z = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;

  // calculate offset
  float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].z * voxel_x + range_min_x;
  float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].y * voxel_y + range_min_y;
  float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].x * voxel_z + range_min_z;

  // feature-offset
  float3 center;
  center.x = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
  center.y = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
  center.z = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

  // store output
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx].x;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx].y;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx].z;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx].w;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;

  } else {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
  }

  __syncthreads();

  for (int i = 0; i < ENCODER_IN_FEATURE_SIZE; i++) {
    int outputSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                     i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int outputId = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                   i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    features[outputId] = ((float *)pillarOutSM)[outputSMId];
  }
}

// cspell: ignore divup
hipError_t generateFeatures_launch(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const std::size_t max_voxel_size, const float voxel_size_x,
  const float voxel_size_y, const float voxel_size_z, const float range_min_x,
  const float range_min_y, const float range_min_z, float * features, hipStream_t stream)
{
  dim3 blocks(divup(max_voxel_size, WARPS_PER_BLOCK));
  dim3 threads(WARPS_PER_BLOCK * MAX_POINT_IN_VOXEL_SIZE);
  generateFeatures_kernel<<<blocks, threads, 0, stream>>>(
    voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
    range_min_x, range_min_y, range_min_z, features);

  return hipGetLastError();
}

}  // namespace autoware::lidar_centerpoint
