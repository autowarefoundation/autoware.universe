#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "autoware/lidar_centerpoint/postprocess/circle_nms_kernel.hpp"
#include "autoware/lidar_centerpoint/postprocess/postprocess_kernel.hpp"

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

namespace
{
const std::size_t THREADS_PER_BLOCK = 32;
}  // namespace

namespace autoware::lidar_centerpoint
{

struct is_score_greater
{
  is_score_greater(float t) : t_(t) {}

  __device__ bool operator()(const Box3D & b) { return b.score > t_; }

private:
  float t_{0.0};
};

struct is_kept
{
  __device__ bool operator()(const bool keep) { return keep; }
};

struct score_greater
{
  __device__ bool operator()(const Box3D & lb, const Box3D & rb) { return lb.score > rb.score; }
};

__device__ inline float sigmoid(float x)
{
  return 1.0f / (1.0f + expf(-x));
}

__global__ void generateBoxes3D_kernel(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, const float voxel_size_x, const float voxel_size_y,
  const float range_min_x, const float range_min_y, const std::size_t down_grid_size_x,
  const std::size_t down_grid_size_y, const std::size_t downsample_factor, const int class_size,
  const bool has_variance, const float * yaw_norm_thresholds, Box3D * det_boxes3d)
{
  // generate boxes3d from the outputs of the network.
  // shape of out_*: (N, DOWN_GRID_SIZE_Y, DOWN_GRID_SIZE_X)
  // heatmap: N = class_size, offset: N = 2, z: N = 1, dim: N = 3, rot: N = 2, vel: N = 2
  const auto yi = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  const auto xi = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const auto idx = down_grid_size_x * yi + xi;
  const auto down_grid_size = down_grid_size_y * down_grid_size_x;

  if (yi >= down_grid_size_y || xi >= down_grid_size_x) {
    return;
  }

  int label = -1;
  float max_score = -1;
  for (int ci = 0; ci < class_size; ci++) {
    float score = sigmoid(out_heatmap[down_grid_size * ci + idx]);
    if (score > max_score) {
      label = ci;
      max_score = score;
    }
  }

  const float offset_x = out_offset[down_grid_size * 0 + idx];
  const float offset_y = out_offset[down_grid_size * 1 + idx];
  const float x = voxel_size_x * downsample_factor * (xi + offset_x) + range_min_x;
  const float y = voxel_size_y * downsample_factor * (yi + offset_y) + range_min_y;
  const float z = out_z[idx];
  const float w = out_dim[down_grid_size * 0 + idx];
  const float l = out_dim[down_grid_size * 1 + idx];
  const float h = out_dim[down_grid_size * 2 + idx];
  const float yaw_sin = out_rot[down_grid_size * 0 + idx];
  const float yaw_cos = out_rot[down_grid_size * 1 + idx];
  const float yaw_norm = sqrtf(yaw_sin * yaw_sin + yaw_cos * yaw_cos);
  const float vel_x = out_vel[down_grid_size * 0 + idx];
  const float vel_y = out_vel[down_grid_size * 1 + idx];

  det_boxes3d[idx].label = label;
  det_boxes3d[idx].score = yaw_norm >= yaw_norm_thresholds[label] ? max_score : 0.f;
  det_boxes3d[idx].x = x;
  det_boxes3d[idx].y = y;
  det_boxes3d[idx].z = z;
  det_boxes3d[idx].length = expf(l);
  det_boxes3d[idx].width = expf(w);
  det_boxes3d[idx].height = expf(h);
  det_boxes3d[idx].yaw = atan2f(yaw_sin, yaw_cos);
  det_boxes3d[idx].vel_x = vel_x;
  det_boxes3d[idx].vel_y = vel_y;

  if (has_variance) {
    const float offset_x_variance = out_offset[down_grid_size * 2 + idx];
    const float offset_y_variance = out_offset[down_grid_size * 3 + idx];
    const float z_variance = out_z[down_grid_size * 1 + idx];
    const float w_variance = out_dim[down_grid_size * 3 + idx];
    const float l_variance = out_dim[down_grid_size * 4 + idx];
    const float h_variance = out_dim[down_grid_size * 5 + idx];
    const float yaw_sin_log_variance = out_rot[down_grid_size * 2 + idx];
    const float yaw_cos_log_variance = out_rot[down_grid_size * 3 + idx];
    const float vel_x_variance = out_vel[down_grid_size * 2 + idx];
    const float vel_y_variance = out_vel[down_grid_size * 3 + idx];

    det_boxes3d[idx].x_variance = voxel_size_x * downsample_factor * expf(offset_x_variance);
    det_boxes3d[idx].y_variance = voxel_size_x * downsample_factor * expf(offset_y_variance);
    det_boxes3d[idx].z_variance = expf(z_variance);
    det_boxes3d[idx].length_variance = expf(l_variance);
    det_boxes3d[idx].width_variance = expf(w_variance);
    det_boxes3d[idx].height_variance = expf(h_variance);
    const float yaw_sin_sq = yaw_sin * yaw_sin;
    const float yaw_cos_sq = yaw_cos * yaw_cos;
    const float yaw_norm_sq = (yaw_sin_sq + yaw_cos_sq) * (yaw_sin_sq + yaw_cos_sq);
    det_boxes3d[idx].yaw_variance =
      (yaw_cos_sq * expf(yaw_sin_log_variance) + yaw_sin_sq * expf(yaw_cos_log_variance)) /
      yaw_norm_sq;
    det_boxes3d[idx].vel_x_variance = expf(vel_x_variance);
    det_boxes3d[idx].vel_y_variance = expf(vel_y_variance);
  }
}

PostProcessCUDA::PostProcessCUDA(const CenterPointConfig & config) : config_(config)
{
}

// cspell: ignore divup
hipError_t PostProcessCUDA::generateDetectedBoxes3D_launch(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, std::vector<Box3D> & det_boxes3d,
  hipStream_t stream)
{
  dim3 blocks(
    divup(config_.down_grid_size_y_, THREADS_PER_BLOCK),
    divup(config_.down_grid_size_x_, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  auto boxes3d_d =
    thrust::device_vector<Box3D>(config_.down_grid_size_y_ * config_.down_grid_size_x_);
  auto yaw_norm_thresholds_d = thrust::device_vector<float>(
    config_.yaw_norm_thresholds_.begin(), config_.yaw_norm_thresholds_.end());
  generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
    out_heatmap, out_offset, out_z, out_dim, out_rot, out_vel, config_.voxel_size_x_,
    config_.voxel_size_y_, config_.range_min_x_, config_.range_min_y_, config_.down_grid_size_x_,
    config_.down_grid_size_y_, config_.downsample_factor_, config_.class_size_,
    config_.has_variance_, thrust::raw_pointer_cast(yaw_norm_thresholds_d.data()),
    thrust::raw_pointer_cast(boxes3d_d.data()));

  // suppress by score
  const auto num_det_boxes3d = thrust::count_if(
    thrust::device, boxes3d_d.begin(), boxes3d_d.end(), is_score_greater(config_.score_threshold_));
  if (num_det_boxes3d == 0) {
    return hipGetLastError();
  }
  thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
  thrust::copy_if(
    thrust::device, boxes3d_d.begin(), boxes3d_d.end(), det_boxes3d_d.begin(),
    is_score_greater(config_.score_threshold_));

  // sort by score
  thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

  // supress by NMS
  thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
  const auto num_final_det_boxes3d =
    circleNMS(det_boxes3d_d, config_.circle_nms_dist_threshold_, final_keep_mask_d, stream);

  thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
  thrust::copy_if(
    thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
    final_det_boxes3d_d.begin(), is_kept());

  // memcpy device to host
  det_boxes3d.resize(num_final_det_boxes3d);
  thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());

  return hipGetLastError();
}

}  // namespace autoware::lidar_centerpoint
