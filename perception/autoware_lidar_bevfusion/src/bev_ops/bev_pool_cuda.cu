// Copyright (c) OpenMMLab. All rights reserved.
// Modified from
// https://github.com/open-mmlab/mmdetection3d/blob/main/projects/BEVFusion/bevfusion/ops/bev_pool/src/bev_pool_cuda.cu
// https://github.com/mit-han-lab/bevfusion/blob/main/mmdet3d/ops/bev_pool/src/bev_pool_cuda.cu
// Available under Apache-2.0 license


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
  Function: pillar pooling
  Args:
    b                : batch size
    d                : depth of the feature map
    h                : height of pooled feature map
    w                : width of pooled feature map
    n                : number of input points
    c                : number of channels
    n_intervals      : number of unique points
    x                : input features, FloatTensor[n, c]
    geom_feats       : input coordinates, IntTensor[n, 4]
    interval_lengths : starting position for pooled point, IntTensor[n_intervals]
    interval_starts  : how many points in each pooled point, IntTensor[n_intervals]
    out              : output features, FloatTensor[b, d, h, w, c]
*/
__global__ void bev_pool_kernel(
  int b, int d, int h, int w, int n, int c, int n_intervals, const float * __restrict__ x,
  const int * __restrict__ geom_feats, const int * __restrict__ interval_starts,
  const int * __restrict__ interval_lengths, float * __restrict__ out)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int index = idx / c;
  int cur_c = idx % c;
  if (index >= n_intervals) return;
  int interval_start = interval_starts[index];
  int interval_length = interval_lengths[index];

  const int * cur_geom_feats = geom_feats + interval_start * 4;
  const float * cur_x = x + interval_start * c + cur_c;
  float * cur_out = out + cur_geom_feats[3] * d * h * w * c + cur_geom_feats[2] * h * w * c +
                    cur_geom_feats[0] * w * c + cur_geom_feats[1] * c + cur_c;
  float psum = 0;
  for (int i = 0; i < interval_length; i++) {
    psum += cur_x[i * c];
  }

  *cur_out = psum;
}

void bev_pool(
  int b, int d, int h, int w, int n, int c, int n_intervals, const float * x,
  const int * geom_feats, const int * interval_starts, const int * interval_lengths, float * out,
  hipStream_t & stream)
{
  bev_pool_kernel<<<(int)ceil(((double)n_intervals * c / 256)), 256, 0, stream>>>(
    b, d, h, w, n, c, n_intervals, x, geom_feats, interval_starts, interval_lengths, out);
}
