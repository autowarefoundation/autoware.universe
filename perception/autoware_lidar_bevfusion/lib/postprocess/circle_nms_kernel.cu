#include "hip/hip_runtime.h"
// Copyright 2025 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// Modified from
// https://github.com/open-mmlab/OpenPCDet/blob/master/pcdet/ops/iou3d_nms/src/iou3d_nms_kernel.cu

/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/

#include "autoware/lidar_bevfusion/postprocess/circle_nms_kernel.hpp"
#include "autoware/lidar_bevfusion/utils.hpp"

#include <autoware/cuda_utils/cuda_check_error.hpp>

#include <thrust/host_vector.h>

#include <cstddef>

namespace
{
const std::size_t THREADS_PER_BLOCK_NMS = 16;
}  // namespace

namespace autoware::lidar_bevfusion
{

__device__ inline float dist2dPow(const Box3D * a, const Box3D * b)
{
  return powf(a->x - b->x, 2) + powf(a->y - b->y, 2);
}

// cspell: ignore divup
__global__ void circleNMS_Kernel(
  const Box3D * __restrict__ boxes, const std::size_t num_boxes3d, const std::size_t col_blocks,
  const float dist2d_pow_threshold, std::uint64_t * __restrict__ mask)
{
  // params: boxes (N,)
  // params: mask (N, divup(N/THREADS_PER_BLOCK_NMS))

  const auto row_start = blockIdx.y;
  const auto col_start = blockIdx.x;

  if (row_start > col_start) return;

  const std::size_t row_size =
    fminf(num_boxes3d - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
  const std::size_t col_size =
    fminf(num_boxes3d - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

  __shared__ Box3D block_boxes[THREADS_PER_BLOCK_NMS];

  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x] = boxes[THREADS_PER_BLOCK_NMS * col_start + threadIdx.x];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const std::size_t cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const Box3D * cur_box = boxes + cur_box_idx;

    std::uint64_t t = 0;
    std::size_t start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (std::size_t i = start; i < col_size; i++) {
      if (dist2dPow(cur_box, block_boxes + i) < dist2d_pow_threshold) {
        t |= 1ULL << i;
      }
    }
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

hipError_t circleNMS_launch(
  const thrust::device_vector<Box3D> & boxes3d, const std::size_t num_boxes3d,
  std::size_t col_blocks, const float distance_threshold,
  thrust::device_vector<std::uint64_t> & mask, hipStream_t stream)
{
  const float dist2d_pow_thres = powf(distance_threshold, 2);

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(THREADS_PER_BLOCK_NMS);
  circleNMS_Kernel<<<blocks, threads, 0, stream>>>(
    thrust::raw_pointer_cast(boxes3d.data()), num_boxes3d, col_blocks, dist2d_pow_thres,
    thrust::raw_pointer_cast(mask.data()));

  return hipGetLastError();
}

std::size_t circleNMS(
  thrust::device_vector<Box3D> & boxes3d, const float distance_threshold,
  thrust::device_vector<bool> & keep_mask, hipStream_t stream)
{
  const auto num_boxes3d = boxes3d.size();
  const auto col_blocks = divup(num_boxes3d, THREADS_PER_BLOCK_NMS);
  thrust::device_vector<std::uint64_t> mask_d(num_boxes3d * col_blocks);

  CHECK_CUDA_ERROR(
    circleNMS_launch(boxes3d, num_boxes3d, col_blocks, distance_threshold, mask_d, stream));

  // memcpy device to host
  thrust::host_vector<std::uint64_t> mask_h(mask_d.size());
  thrust::copy(mask_d.begin(), mask_d.end(), mask_h.begin());
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

  // generate keep_mask
  std::vector<std::uint64_t> remv_h(col_blocks);
  thrust::host_vector<bool> keep_mask_h(keep_mask.size());
  std::size_t num_to_keep = 0;
  for (std::size_t i = 0; i < num_boxes3d; i++) {
    auto nblock = i / THREADS_PER_BLOCK_NMS;
    auto inblock = i % THREADS_PER_BLOCK_NMS;

    if (!(remv_h[nblock] & (1ULL << inblock))) {
      keep_mask_h[i] = true;
      num_to_keep++;
      std::uint64_t * p = &mask_h[0] + i * col_blocks;
      for (std::size_t j = nblock; j < col_blocks; j++) {
        remv_h[j] |= p[j];
      }
    } else {
      keep_mask_h[i] = false;
    }
  }

  // memcpy host to device
  keep_mask = keep_mask_h;

  return num_to_keep;
}

}  // namespace autoware::lidar_bevfusion
